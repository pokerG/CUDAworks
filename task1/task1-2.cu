
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b){
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main(){
	const int arraySize = 3000;
	int i = 0;
	int a[arraySize];
	int b[arraySize];
	int c[arraySize] = { 0 };
	for(i = 0; i < arraySize; i++){
		a[i] = (int)(rand()% 100);
		b[i] = (int)(rand()% 100);
	}

	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda  failed!");
		return 1;
	}

	for(i = 0;i < arraySize; i++){
		printf("%d + %d = %d\t",a[i],b[i],c[i]);
	}


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size){
	hipError_t cudaStatus;
   
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;

}