#include <stdio.h>
#include "hip/hip_runtime.h"

#include "string.h"

void printDevice(hipDeviceProp_t prop){
	printf("\t Name: \t%s\n",prop.name);
    printf("\t Capability Major/Minor version number:    %d.%d\n", prop.major, prop.minor);
	printf("\t Total amount of global memory: \t%.0f MBytes (%llu bytes)\n",
                (float)prop.totalGlobalMem/1048576.0f, (unsigned long long) prop.totalGlobalMem);
	printf("\t maxThreadsPerBlock: \t%d\n",prop.maxThreadsPerBlock);
	printf("\t totalConstMen: \t%d\n",prop.totalConstMem);
	printf("\t sharedMemPerBlcok: \t%d\n",prop.sharedMemPerBlock);
	printf("\t regsPerBlock: \t%d\n",prop.regsPerBlock);
	printf("\t maxThreadsPerMultiProcessor: \t%d\n",prop.maxThreadsPerMultiProcessor);
	printf("\t multiProcessorCount: \t%d\n",prop.multiProcessorCount);
}

int main(){

	int count;
	hipGetDeviceCount(&count);
	printf("There are %d devices.\n",count);	
	int i;
	for(i = 0; i < count; i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i) == hipSuccess){
			printf("The %dth device's informations\n",i + 1);
			printDevice(prop);
		}

	}

	return 0;
}