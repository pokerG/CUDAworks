#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void whatIsMyId (unsigned int * const block,unsigned int * const thread,unsigned int * const warp,unsigned int * const clacThread){
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	block[idx] = blockIdx.x;
	thread[idx] = threadIdx.x;
	warp[idx] = threadIdx.x / warpSize;
	clacThread[idx] = idx;
}
#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
unsigned int cpuBlock[ARRAY_SIZE];
unsigned int cpuThread[ARRAY_SIZE];
unsigned int cpuWarp[ARRAY_SIZE];
unsigned int cpucalcThread[ARRAY_SIZE];

int main(){
	const unsigned int numBlock = 2;
	const unsigned int numThreads = 64;
	char ch;

	unsigned int * gpuBlock;
	unsigned int * gpuThread;
	unsigned int * gpuWarp;
	unsigned int * gpuclacThread;
	unsigned int i;

	hipMalloc((void **)&gpuBlock,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuThread,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuWarp,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuclacThread,ARRAY_SIZE_IN_BYTES);

	whatIsMyId<<<numBlock,numThreads>>>(gpuBlock,gpuThread,gpuWarp,gpuclacThread);

	hipMemcpy(cpuBlock,gpuBlock,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuThread,gpuThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuWarp,gpuWarp,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpucalcThread,gpuclacThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);

	hipFree(gpuBlock);
	hipFree(gpuThread);
	hipFree(gpuWarp);
	hipFree(gpuclacThread);

	for(i = 0; i < ARRAY_SIZE; i++){
		printf("Calculate Thread: %3u - Block�� %2u - Warp %2u - Thread: %3u\n",
			cpucalcThread[i],cpuBlock[i],cpuWarp[i],cpuThread[i]);
	}
	ch = getch();
}