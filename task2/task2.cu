#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>


__global__ void whatIsMyId (unsigned int * const block,unsigned int * const thread,unsigned int * const warp,unsigned int * const clacThread, float * const tm){
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	block[idx] = blockIdx.x;
	thread[idx] = threadIdx.x;
	warp[idx] = threadIdx.x / warpSize;
	clacThread[idx] = idx;
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tm[idx],start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
#define ARRAY_SIZE_IN_FLOAT (sizeof(float) * ARRAY_SIZE)
unsigned int cpuBlock[ARRAY_SIZE];
unsigned int cpuThread[ARRAY_SIZE];
unsigned int cpuWarp[ARRAY_SIZE];
unsigned int cpucalcThread[ARRAY_SIZE];

float cpuTime[ARRAY_SIZE];


int main(){
	const unsigned int numBlock = 2;
	const unsigned int numThreads = 64;
	char ch;

	unsigned int * gpuBlock;
	unsigned int * gpuThread;
	unsigned int * gpuWarp;
	unsigned int * gpuclacThread;
	unsigned int i;
	float *gpuTime;

	hipMalloc((void **)&gpuBlock,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuThread,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuWarp,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuclacThread,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuTime,ARRAY_SIZE_IN_FLOAT);

	whatIsMyId<<<numBlock,numThreads>>>(gpuBlock,gpuThread,gpuWarp,gpuclacThread,gpuTime);

	hipMemcpy(cpuBlock,gpuBlock,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuThread,gpuThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuWarp,gpuWarp,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpucalcThread,gpuclacThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuTime,gpuTime,ARRAY_SIZE_IN_FLOAT,hipMemcpyDeviceToHost);

	hipFree(gpuTime);
	hipFree(gpuBlock);
	hipFree(gpuThread);
	hipFree(gpuWarp);
	hipFree(gpuclacThread);

	hipDeviceProp_t prop;
	if(hipGetDeviceProperties(&prop,0) == hipSuccess){
		printf("warpSize: %d\n",prop.warpSize);
	}
	for(i = 0; i < ARRAY_SIZE; i++){
		printf("Calculate Thread: %3u - Block�� %2u - Warp %2u - Thread: %3u\n",
			cpucalcThread[i],cpuBlock[i],cpuWarp[i],cpuThread[i]);
	}
	ch = getch();
}