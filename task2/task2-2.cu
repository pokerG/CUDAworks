#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>

__global__ void whatIsMyId (unsigned int * const block,unsigned int * const thread,unsigned int * const warp,unsigned int * const clacThread,clock_t * timer){
	

	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(idx % warpSize == 0) timer[idx / warpSize] = clock();	
	block[idx] = blockIdx.x;
	thread[idx] = threadIdx.x;
	warp[idx] = threadIdx.x / warpSize;
	clacThread[idx] = idx;
	if(idx % warpSize == 0) timer[gridDim.x * 2 + idx / warpSize] = clock();
	
}
#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
#define ARRAY_SIZE_IN_FLOAT (sizeof(float) * ARRAY_SIZE)
unsigned int cpuBlock[ARRAY_SIZE];
unsigned int cpuThread[ARRAY_SIZE];
unsigned int cpuWarp[ARRAY_SIZE];
unsigned int cpucalcThread[ARRAY_SIZE];

//float cpuTime[ARRAY_SIZE];


int main(){
	const unsigned int numBlock = 65;
	const unsigned int numThreads = 64;
	char ch;

	unsigned int * gpuBlock;
	unsigned int * gpuThread;
	unsigned int * gpuWarp;
	unsigned int * gpuclacThread;
	unsigned int i;
	//float *gpuTime;
	clock_t *gpuTime;
	clock_t cpuTime[numBlock * 4];
	
	hipMalloc((void **)&gpuTime,sizeof(clock_t) * numBlock * 4);
	hipMalloc((void **)&gpuBlock,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuThread,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuWarp,ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpuclacThread,ARRAY_SIZE_IN_BYTES);
	//hipMalloc((void **)&gpuTime,ARRAY_SIZE_IN_FLOAT);

//	hipEvent_t start,stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start,0);
	whatIsMyId<<<numBlock,numThreads>>>(gpuBlock,gpuThread,gpuWarp,gpuclacThread,gpuTime);
//	hipEventRecord(stop,0);
//	hipEventSynchronize(stop);
//	float elapsedTime;
//	hipEventElapsedTime(&elapsedTime,start,stop);
//	hipEventDestroy(start);
//	hipEventDestroy(stop);

	hipMemcpy(cpuTime,gpuTime,sizeof(clock_t) * numBlock * 4,hipMemcpyDeviceToHost);
	hipMemcpy(cpuBlock,gpuBlock,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuThread,gpuThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpuWarp,gpuWarp,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	hipMemcpy(cpucalcThread,gpuclacThread,ARRAY_SIZE_IN_BYTES,hipMemcpyDeviceToHost);
	//hipMemcpy(cpuTime,gpuTime,ARRAY_SIZE_IN_FLOAT,hipMemcpyDeviceToHost);

	//hipFree(gpuTime);
	hipFree(gpuTime);
	hipFree(gpuBlock);
	hipFree(gpuThread);
	hipFree(gpuWarp);
	hipFree(gpuclacThread);

	hipDeviceProp_t prop;
	if(hipGetDeviceProperties(&prop,0) == hipSuccess){
		printf("warpSize: %d\n",prop.warpSize);
	}
	for(i = 0; i < ARRAY_SIZE; i++){
		printf("Calculate Thread: %3u - Block�� %2u - Warp %2u - Thread: %3u\n",
			cpucalcThread[i],cpuBlock[i],cpuWarp[i],cpuThread[i]);
	}

	for(i = 0; i < numBlock * 2; i++){
		//printf("%d %d\n",cpuTime[numBlock * 2 + i],cpuTime[i]);
		printf("warp %d time = %d\n",i,cpuTime[numBlock * 2 + i]-cpuTime[i]);
	}
	
	ch = getch();

}