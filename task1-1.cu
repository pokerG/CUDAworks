#include <stdio.h>
#include "hip/hip_runtime.h"

#include "string.h"

/*bool InitCUDA(){
	int count;
	cudaGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.]n");
		return false;
	}
	int i;
	for(i = 0; i < count; i++){
		cudaDeviceProp prop;
		if(cudaGetDeviceProperties(&prop,i) == cudaSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	}
	if(i == count){
		fprintf(stderr,"There is no device suppoting CUDA 1.x.\n");
		return false;
	}
	cudaSetDevice(i);
	return true;
}*/
void printDevice(hipDeviceProp_t prop){
	printf("\t Name: \t%s\n",prop.name);
	printf("\t totalGlobalMen: \t%ld\n",prop.totalGlobalMem);
	printf("\t maxThreadsPerBlock: \t%d\n",prop.maxThreadsPerBlock);
	printf("\t totalConstMen: \t%d\n",prop.totalConstMem);
	printf("\t sharedMemPerBlcok: \t%d\n",prop.sharedMemPerBlock);
	printf("\t regsPerBlock: \t%d\n",prop.regsPerBlock);
	printf("\t maxThreadsPerMultiProcessor: \t%d\n",prop.maxThreadsPerMultiProcessor);
	printf("\t multiProcessorCount: \t%d\n",prop.multiProcessorCount);
}

int main(){

	int count;
	hipGetDeviceCount(&count);
	printf("There are %d devices.\n",count);	
	int i;
	for(i = 0; i < count; i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i) == hipSuccess){
			printf("The %dth device's informations\n",i + 1);
			printDevice(prop);
		}

	}

	return 0;
}