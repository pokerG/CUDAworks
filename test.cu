#include <stdio.h>
#include "hip/hip_runtime.h"


/*bool InitCUDA(){
	int count;
	cudaGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.]n");
		return false;
	}
	int i;
	for(i = 0; i < count; i++){
		cudaDeviceProp prop;
		if(cudaGetDeviceProperties(&prop,i) == cudaSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	}
	if(i == count){
		fprintf(stderr,"There is no device suppoting CUDA 1.x.\n");
		return false;
	}
	cudaSetDevice(i);
	return true;
}*/

int main(){

	int count;
	hipGetDeviceCount(&count);
	printf("There are %d devices.\n",count);


	return 0;
}